#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "pyutils/pyutils.cuh"
using namespace kittens;

using my_layout = gl<float, -1, -1, -1, 64, st_fl<64, 64>>;  // An example layout that also instantiates a TMA descriptor on Hopper.
struct globals {
    my_layout in, out;
    dim3 grid() { return dim3(in.batch(), in.depth(), in.rows()); }
    dim3 block() { return dim3(in.cols()); }
};
__global__ void copy_kernel(const __grid_constant__ globals g) {
    if (threadIdx.x == 0 && blockIdx.x == 0 &&
        blockIdx.y == 0 && blockIdx.z == 0)
        printf("Hello, from inside the kernel!\n");

    // Cast CUDA built-ins to avoid narrowing warnings
    const int bx = static_cast<int>(blockIdx.x);
    const int by = static_cast<int>(blockIdx.y);
    const int bz = static_cast<int>(blockIdx.z);
    const int tx = static_cast<int>(threadIdx.x);

    g.out[{bx, by, bz, tx}] = g.in[{bx, by, bz, tx}];
}
void run_copy_kernel(globals g) {
    printf("I am calling the kernel from the host.\n");
    copy_kernel<<<g.grid(), g.block()>>>(g);
}

PYBIND11_MODULE(example_bind, m) {
    m.doc() = "example_bind python module";
    py::bind_kernel<copy_kernel>(m, "copy_kernel", &globals::in, &globals::out);
    py::bind_function<run_copy_kernel>(m, "wrapped_copy_kernel", &globals::in, &globals::out);
}
